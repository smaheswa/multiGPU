#include <iostream>


void printDeviceProperties(){

	int devID;
	hipDeviceProp_t props;

	hipGetDevice(&devID);
	hipGetDeviceProperties(&props, devID);
	std::cout<< 
		" Device: " << devID << "\n" << 
		" with compute: " << props.name << "\n" <<
		" capability: " <<props.name << "." << props.minor << "\n" <<  
		" totalGlobalMemory: " << props.totalGlobalMem << "\n" << 
		" sharedMemPerBlock: " << props.sharedMemPerBlock << "\n" <<
		" maxThreadsPerBlock: " << props.maxThreadsPerBlock << "\n" <<
		" totalConstantMemory: " << props.totalConstMem << "\n" << 
		" compute Mode: " << props.computeMode << "\n" << 
		" ECC enabled: " << props.ECCEnabled << std::endl;

}
