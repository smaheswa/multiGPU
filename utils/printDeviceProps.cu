#include <iostream>


void printDeviceProperties(){

	int devID;
	hipDeviceProp_t props;

	hipGetDevice(&devID);
	hipGetDeviceProperties(&props, devID);
	std::cout<< 
		"\tDevice: " << devID << "\n" << 
		"\twith compute: " << props.name << "\n" <<
		"\tcapability: " <<props.name << "." << props.minor << "\n" <<  
		"\ttotalGlobalMemory: " << props.totalGlobalMem << "\n" << 
		"\tsharedMemPerBlock: " << props.sharedMemPerBlock << "\n" <<
		"\tmaxThreadsPerBlock: " << props.maxThreadsPerBlock << "\n" <<
		"\totalConstantMemory: " << props.totalConstMem << "\n" << 
		"\tcompute Mode: " << props.computeMode << "\n" << 
		"\tECC enabled: " << props.ECCEnabled << std::endl;
}
