#include <stdio.h>
#include <iostream>


void testSimplePrint(){

	int devID;
	hipDeviceProp_t props;

	hipGetDevice(&devID);
	hipGetDeviceProperties(&props, devID);
	std::cout<< 
		" Device: " << devID << "\n" << 
		" with compute: " << props.name << "\n" <<
		" capability: " <<props.name << "." << props.minor << "\n" <<  
		" totalGlobalMemory: " << props.totalGlobalMem << "\n" << 
		" sharedMemPerBlock: " << props.sharedMemPerBlock << "\n" <<
		" maxThreadsPerBlock: " << props.maxThreadsPerBlock << "\n" <<
		" totalConstantMemory: " << props.totalConstMem << "\n" << 
		" compute Mode: " << props.computeMode << "\n" << 
		" ECC enabled: " << props.ECCEnabled << std::endl;

	//printf("Device %d: \"%s\" with compute %d.%d capability\n",
//			devID, props.name, props.major, props.minor)
}
int main(int argc, char **argv) {

	testSimplePrint();
	return 0;
}
